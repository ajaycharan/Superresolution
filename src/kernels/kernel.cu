
#include <hip/hip_runtime.h>
///*
// * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
// *
// * Please refer to the NVIDIA end user license agreement (EULA) associated
// * with this source code for terms and conditions that govern your use of
// * this software. Any use, reproduction, disclosure, or distribution of
// * this software and related documentation outside the terms of the EULA
// * is strictly prohibited.
// *
// */

// /* This example demonstrates how to use the Cuda OpenGL bindings with the
//  * runtime API.
//  * Device code.
//  */

//#ifndef _SIMPLEGL_KERNEL_H_
//#define _SIMPLEGL_KERNEL_H_

//#include <stdio.h>
//#include <cutil_inline.h>
//#ifndef max
//#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
//#endif

//#ifndef min
//#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
//#endif


///////////////////////////////////////////////////////////////////////////////////
//////! Simple kernel to modify vertex positions in sine wave pattern
//////! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////////

////__global__ void kernel(int *d_Darray, int size)
////{
//////    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//////    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

////    // write output vertex

//////    printf("d_Darray[%d]= %d",threadIdx.x,d_Darray[threadIdx.x]);
////}


//__global__ void kernel_dualp(float *px, float *py, float *ux_, float *uy_, float sigma, unsigned int stride, unsigned int width, unsigned int height)
//{


//    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    // write output vertex
//    px[y*stride+x] = px[y*stride+x] + sigma* ux_[y*stride+x];
//    py[y*stride+x] = py[y*stride+x] + sigma* uy_[y*stride+x];

//    float pxval = px[y*stride+x];
//    float pyval = py[y*stride+x];

//    float reprojection = 0;
//    reprojection   = sqrt(pxval*pxval + pyval*pyval);
//    reprojection   = max(1,reprojection);

//    px[y*stride+x] = px[y*stride+x]/reprojection;
//    py[y*stride+x] = py[y*stride+x]/reprojection;


//}


//__global__ void kernel_update_u(float *px, float *py, float *u, float *u_, float* g ,unsigned int stride, unsigned int width, unsigned int height, float tau, float lambda)
//{

//    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    float dxp = 0 , dyp = 0;

//    if ( x >= 1 && x < width )  dxp = px[y*stride+x] - px[y*stride+(x-1)];

//    if ( y >= 1 && y < height ) dyp = py[y*stride+x] - py[(y-1)*stride+x];

//    float divp = dxp + dyp;

//    float u_prev = u[y*stride+x];

//    u[y*stride+x] = (u_prev + tau*divp+ tau*lambda*g[y*stride+x])/(1+tau*lambda);

//    u_[y*stride+x] = 2*u[y*stride+x] - u_prev;
//   //  u_[y*stride+x] = u[y*stride+x];// - u_prev;


//}


////extern "C" void launch_kernel(int *d_Darray, int size)
////{
////    dim3 block(size,1,1);
////    dim3 grid(1,1);
////    kernel<<< grid, block>>>(d_Darray,size);
////    cutilCheckMsg("execution failed\n");
////}




//#endif // #ifndef _SIMPLEGL_KERNEL_H_

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /* This example demonstrates how to use the Cuda OpenGL bindings with the
  * runtime API.
  * Device code.
  */

#ifndef _SIMPLEGL_KERNEL_H_
#define _SIMPLEGL_KERNEL_H_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


/////////////////////////////////////////////////////////////////////////////////
////! Simple kernel to modify vertex positions in sine wave pattern
////! @param data  data in global memory
/////////////////////////////////////////////////////////////////////////////////

//__global__ void kernel(int *d_Darray, int size)
//{
////    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
////    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    // write output vertex

////    printf("d_Darray[%d]= %d",threadIdx.x,d_Darray[threadIdx.x]);
//}


__global__ void kernel_dualp(float *px, float *py, float *ux_, float *uy_, float sigma, unsigned int stride, unsigned int width, unsigned int height)
{


    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    px[y*stride+x] = px[y*stride+x] + sigma* ux_[y*stride+x];
    py[y*stride+x] = py[y*stride+x] + sigma* uy_[y*stride+x];

    float pxval = px[y*stride+x];
    float pyval = py[y*stride+x];

    float reprojection = 0;
    reprojection   = sqrt(pxval*pxval + pyval*pyval);
    reprojection   = max(1,reprojection);

    px[y*stride+x] = px[y*stride+x]/reprojection;
    py[y*stride+x] = py[y*stride+x]/reprojection;


}


__global__ void kernel_update_u(float *px, float *py, float *u, float *u_, float* g ,unsigned int stride, unsigned int width, unsigned int height, float tau, float lambda)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float dxp = 0 , dyp = 0;

    if ( x >= 1 && x < width )  dxp = px[y*stride+x] - px[y*stride+(x-1)];

    if ( y >= 1 && y < height ) dyp = py[y*stride+x] - py[(y-1)*stride+x];

    float divp = dxp + dyp;

    float u_prev = u[y*stride+x];

    u[y*stride+x] = (u_prev + tau*divp+ tau*lambda*g[y*stride+x])/(1+tau*lambda);

    u_[y*stride+x] = 2*u[y*stride+x] - u_prev;
   //  u_[y*stride+x] = u[y*stride+x];// - u_prev;


}


__global__ void kernel_derivative_u(float *ux_, float *uy_, float *u_, unsigned int stride, unsigned int width, unsigned int height )
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x + 1 < width ) //&& y < height )
    {
        ux_[y*stride+x] = u_[y*stride+(x+1)] - u_[y*stride+x];
    }

    if ( y + 1 < height )
    {
        uy_[y*stride+x] = u_[(y+1)*stride+x] - u_[y*stride+x];
    }

}

extern "C" void launch_kernel_derivative_u(float* ux_, float *uy_, float* u_, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);


    kernel_derivative_u<<< grid, block>>>(ux_, uy_, u_, stride, mesh_width, mesh_height);


    cutilCheckMsg("execution failed\n");


}


extern "C" void launch_kernel_update_u(float *px, float *py, float *u, float *u_, float* g, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height, float tau, float lambda)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_update_u<<< grid, block>>>(px,py,u, u_, g, stride, mesh_width, mesh_height, tau, lambda);
    cutilCheckMsg("execution failed\n");
}


// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_dual_variable_p(float *px, float *py, float* ux_, float *uy_, float sigma, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_dualp<<< grid, block>>>(px,py,ux_,uy_,sigma, stride, mesh_width, mesh_height);
    cutilCheckMsg("execution failed\n");
}

//extern "C" void launch_kernel(int *d_Darray, int size)
//{
//    dim3 block(size,1,1);
//    dim3 grid(1,1);
//    kernel<<< grid, block>>>(d_Darray,size);
//    cutilCheckMsg("execution failed\n");
//}




#endif // #ifndef _SIMPLEGL_KERNEL_H_

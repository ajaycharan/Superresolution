#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_U_
#define _KERNEL_UPDATE_U_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_derivative_u(float *ux_, float *uy_, float *u_, unsigned int stride, unsigned int width, unsigned int height )
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x + 1 < width )
    {
        ux_[y*width+x] = u_[y*width+(x+1)] - u_[y*width+x];
    }

    if ( y + 1 < height )
    {
        uy_[y*width+x] = u_[(y+1)*width+x] - u_[y*width+x];
    }


}

extern "C" void launch_kernel_derivative_u(float* ux_, float *uy_, float* u_, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(1, 1, 1);
    dim3 grid(mesh_width / block.x, (mesh_height / block.y), 1);
    kernel_derivative_u<<< grid, block>>>(ux_, uy_, u_, stride, mesh_width, mesh_height);
    cutilCheckMsg("execution failed\n");

}


//extern "C" void launch_kernel_update_u(float *px, float *py, float *u, float *u_, float* g, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height, float tau, float lambda)
//{
//    // execute the kernel
//    dim3 block(8, 8, 1);
//    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
//    kernel_update_u<<< grid, block>>>(px,py,u, u_, g, stride, mesh_width, mesh_height, tau, lambda);
//    cutilCheckMsg("execution failed\n");
//}


//launch_kernel_primal_u(d_px,d_py,d_u_,d_u, superresolutionImageStride, epsilon_u,tau,xisqr, d_dual_save_WTBTDTq, WTBTDTqStrideFloat,width_up,height_up,N_imgs);

__global__ void kernel_primalu(float *px, float *py, float* u_, float *u, int upImageStrideFloat, float epsilon_u, float* d_tau, float xisqr, float *WiT_BiT_DiT_qi,
                               unsigned int WTBTDTstride, unsigned int width_up, unsigned int height_up, int N_imgs)
{

    //u^{n+1} = u^{n} - tau( -divp^{n+1} + \xi^{2} \sum_{i=1}^{N}(W^{T}B^{T}D^{T}q^{n+1} )

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    if ( y*width_up+x < width_up*height_up)
    {

        float dxp = 0 , dyp = 0;

        if ( x >= 1 && x < width_up )  dxp = px[y*width_up+x] - px[y*width_up+(x-1)];

        if ( y >= 1 && y < height_up ) dyp = py[y*width_up+x] - py[(y-1)*width_up+x];

        float divp = dxp + dyp;

        float prev_u = u[y*width_up+x];

        float tau = d_tau[y*width_up+x];

        u[y*width_up+x]  = prev_u - tau*( -divp + WiT_BiT_DiT_qi[y*width_up+x]);

        u_[y*width_up+x] = 2*u[y*width_up+x] - prev_u;
    }

}


// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_primal_u(float *px, float *py, float* u_, float *u, int upImageStrideFloat, float epsilon_u, float* d_tau, float xisqr, float *WiT_BiT_DiT_qi,
                                       unsigned int WTBTDTstride, unsigned int width_up, unsigned int height_up, int N_imgs)
{
    // execute the kernel
    dim3 block(1, 1, 1);
    dim3 grid(width_up / block.x, height_up / block.y, 1);
    kernel_primalu<<< grid, block>>>(px, py, u_,u, upImageStrideFloat, epsilon_u, d_tau, xisqr, WiT_BiT_DiT_qi, WTBTDTstride, width_up, height_up,N_imgs);
    cutilCheckMsg("execution failed\n");
}

#endif // #ifndef _SIMPLEGL_KERNEL_H_

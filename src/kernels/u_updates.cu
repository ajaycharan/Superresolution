#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_U_
#define _KERNEL_UPDATE_U_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_derivative_u(float *ux_, float *uy_, float *u_, unsigned int stride, unsigned int width, unsigned int height )
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x + 1 < width ) //&& y < height )
    {
        ux_[y*stride+x] = u_[y*stride+(x+1)] - u_[y*stride+x];
    }

    if ( y + 1 < height )
    {
        uy_[y*stride+x] = u_[(y+1)*stride+x] - u_[y*stride+x];
    }

}

extern "C" void launch_kernel_derivative_u(float* ux_, float *uy_, float* u_, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_derivative_u<<< grid, block>>>(ux_, uy_, u_, stride, mesh_width, mesh_height);
    cutilCheckMsg("execution failed\n");

}


//extern "C" void launch_kernel_update_u(float *px, float *py, float *u, float *u_, float* g, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height, float tau, float lambda)
//{
//    // execute the kernel
//    dim3 block(8, 8, 1);
//    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
//    kernel_update_u<<< grid, block>>>(px,py,u, u_, g, stride, mesh_width, mesh_height, tau, lambda);
//    cutilCheckMsg("execution failed\n");
//}



__global__ void kernel_primalu(float *px, float *py, float* u_, float *u, float epsilon_u, float tau, float xisqr, float *WiT_BiT_DiT_qi,
                               unsigned int stride, unsigned int width_up, unsigned int height_up, int N_imgs)
{

    //u^{n+1} = u^{n} - tau( -divp^{n+1} + \xi^{2} \sum_{i=1}^{N}(W^{T}B^{T}D^{T}q^{n+1} )

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex



    float dxp = 0 , dyp = 0;

    if ( x >= 1 && x < width_up )  dxp = px[y*stride+x] - px[y*stride+(x-1)];

    if ( y >= 1 && y < height_up ) dyp = py[y*stride+x] - py[(y-1)*stride+x];

    float divp = dxp + dyp;

    float prev_u = u[y*stride+x];

    int image_size = width_up*height_up;

    float sum_WiT_BiT_DiT_qi = 0;

    for(int i = 0 ; i < N_imgs; i++)
    {
        sum_WiT_BiT_DiT_qi += WiT_BiT_DiT_qi[(y*stride+x)+(image_size)*i];
    }


    u[y*stride+x] = prev_u - tau*( -divp + xisqr*sum_WiT_BiT_DiT_qi);

    u_[y*stride+x] = 2*u[y*stride+x] - prev_u;

}


// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_primal_u(float *px, float *py, float* u_, float *u, float epsilon_u, float tau, float xisqr, float *WiT_BiT_DiT_qi,
                                       unsigned int stride, unsigned int width_up, unsigned int height_up, int N_imgs)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(width_up / block.x, height_up / block.y, 1);
    kernel_primalu<<< grid, block>>>(px, py, u_,u,epsilon_u, tau,xisqr, WiT_BiT_DiT_qi, stride, width_up, height_up,N_imgs);
    cutilCheckMsg("execution failed\n");
}

#endif // #ifndef _SIMPLEGL_KERNEL_H_

#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_U_
#define _KERNEL_UPDATE_U_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_dualu(float tau, float *px, float *py, float xisqr, float *sum_wiT_biT_diT_q, int width, int height,
                             unsigned int stride)
{

    //u^{n+1} = u^{n} - tau( -divp^{n+1} + \xi^{2} \sum_{i=1}^{N}(W^{T}B^{T}D^{T}q^{n+1} )

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex

    float dxp = 0 , dyp = 0;

    if ( x >= 1 && x < width )  dxp = px[y*stride+x] - px[y*stride+(x-1)];

    if ( y >= 1 && y < height ) dyp = py[y*stride+x] - py[(y-1)*stride+x];

    float divp = dxp + dyp;

    float prev_u = u[y*stride+x];

    u[y*stride+x] = prev_u - tau*( -divp + xisqr*sum_wiT_biT_diT_q[y*stride+x]);

    u[y*stride+x] = 2*u[y*stride+x] - prev;

}


// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_dual_variable_u(float *px, float *py, float* ux_, float *uy_, float epsilon_u, float sigma, float lambda,
                                              unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_dualp<<< grid, block>>>(tau, px, py, xisqr, sum_wiT_biT_diT_q, width, height, stride);
    cutilCheckMsg("execution failed\n");
}

#endif // #ifndef _SIMPLEGL_KERNEL_H_

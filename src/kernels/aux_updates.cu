#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_P_
#define _KERNEL_UPDATE_P_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_Mult_Wi_u(float *out, int outStride, float *d_u_, int u_Stride, float *horizontal_flow, float* vertical_flow, int velStride,
                                 int imgNo, int width, int height)
{


    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float x_ = x;
    float y_ = y;

    x_ = min(width-1,max(0.0f,x_ + horizontal_flow[y*velStride+x]));
    y_ = min(height-1,max(0.0f,y_ + vertical_flow[y*velStride+x]));

    int flr_x = floor(x_);
    int flr_y = floor(y_);

    float x_ratio = x_ - flr_x;
    float y_ratio = y_ - flr_y;

    float val = ((1-x_ratio)*(1-y_ratio))*d_u_[flr_y*u_Stride+flr_x];

    if ( flr_x + 1 < width )
        val +=  (x_ratio*(1-y_ratio))*d_u_[flr_y*u_Stride + (flr_x+1)];

    if ( flr_y + 1 < height )
    {
        val +=  ((1-x_ratio)*y_ratio)*d_u_[(flr_y+1)*u_Stride + (flr_x)];

        if ( flr_x+1 < width )
            val +=  (x_ratio*y_ratio)*d_u_[(flr_y+1)*u_Stride + (flr_x+1)];
    }

    out[y*outStride+x] = val;
}



// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_Mult_Wi_u(float *out, int outStride, float *d_u_, int u_Stride, float *horizontal_flow, float* vertical_flow, int velStride,
                                         int imgNo, int width, int height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);
    kernel_Mult_Wi_u<<< grid, block>>>(out, outStride, d_u_, u_Stride,  horizontal_flow, vertical_flow, velStride, imgNo, width, height);
    cutilCheckMsg("execution failed\n");
}


__global__ void kernel_blur(float *out, int outStride, float *in, int inStride, float* blur_kernel, int blurWidth, int width, int height)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float val = 0;

    float sum_kernel = 0;


    for(int i = -blurWidth/2 ; i <= blurWidth/2 ; i++ )
    {
        for(int j = -blurWidth/2 ; j <= blurWidth/2 ; j++ )
        {
            if ( y+i < height && x+j < width )
            {
                val += blur_kernel[i*blurWidth+j]*in[(y+i)*inStride+(x+j)];
                sum_kernel += blur_kernel[i*blurWidth+j];
            }
        }
    }

    out[y*outStride+x] = (val/sum_kernel);

}


__global__ void kernel_subtract(float* d_fi, int imgVectorsStrideFloat, float* d_res_stacked, int qVectorsStrideFloat, int size)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( y*qVectorsStrideFloat+x < size)
    {
        d_res_stacked[y*qVectorsStrideFloat+x] = d_res_stacked[y*qVectorsStrideFloat+x]-d_fi[y*qVectorsStrideFloat+x];
    }

}



extern "C" void launch_kernel_blur(float *out, int outStride, float *in, int inStride, float* blur_kernel, int blurWidth, int width, int height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);
    kernel_blur<<< grid, block>>>(out, outStride, in, inStride, blur_kernel, blurWidth, width, height);
    cutilCheckMsg("execution failed\n");
}

extern "C" void  launch_kernel_subtract(float* d_fi, int imgVectorsStrideFloat, float* d_res_stacked, int qVectorsStrideFloat, int size, int width, int height)
{

    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);
    kernel_subtract<<< grid, block>>>(d_fi, imgVectorsStrideFloat, d_res_stacked, qVectorsStrideFloat, size);
    cutilCheckMsg("execution failed\n");

}


//__global__ void launch_kernel_blurTranspose(float *out, int outStride, float *in, int inStride, float* blur_kernelT, int blurWidth)
//{

//    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    float val = 0;



//    for(int i = -blurWidth/2 ; i <= blurWidth/2 ; i++ )
//    {
//        for(int j = -blurWidth/2 ; j <= blurWidth/2 ; j++ )
//        {
//            if ( y+i < Ny && x+j < Nx )
//            {
//                val += blur_kernel[i*blurWidth+j]*in[(y+i)*inStride+(x+j)];
//            }
//        }
//    }

//    out[y*outStride+x] = val;

//}

//extern "C" void launch_kernel_blurTranspose(float *out, int outStride, float *in, int inStride, float* blur_kernel, int blurWidth)
//{
//    // execute the kernel
//    dim3 block(8, 8, 1);
//    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
//    kernel_blur_Wi_u<<< grid, block>>>(out, outStide, in, inStride, blur_kernel, blurWidth);
//    cutilCheckMsg("execution failed\n");
//}



#endif // #ifndef _SIMPLEGL_KERNEL_H_

#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_P_
#define _KERNEL_UPDATE_P_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_Mult_Wi_u(float *u, float *horizontal_flow, float* vertical_flow, int imgNo,
                                 unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{


    //p^{n+1} = \frac{p^n + \sigma \nabla u^{n}}{1 + sigma*epsilon_u/ lambda}
    //p^{n+1} = p^{n+1} / max ( 1, |p^{n+1}|/lambda )

//    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    // write output vertex
//    px[y*stride+x] = (px[y*stride+x] + sigma* ux_[y*stride+x])/(1+sigma*epsilon_u/lambda);
//    py[y*stride+x] = (py[y*stride+x] + sigma* uy_[y*stride+x])/(1+sigma*epsilon_u/lambda);

//    float pxval = px[y*stride+x];
//    float pyval = py[y*stride+x];

//    float reprojection = 0;
//    reprojection   = sqrt(pxval*pxval + pyval*pyval);
//    reprojection   = max(1,reprojection/lambda);

//    px[y*stride+x] = px[y*stride+x]/reprojection;
//    py[y*stride+x] = py[y*stride+x]/reprojection;


}



// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_Mult_Wi_u(float *u, float *horizontal_flow, float* vertical_flow, int imgNo,
                                              unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_Mult_Wi_u<<< grid, block>>>(u, horizontal_flow, vertical_flow, imgNo, stride, mesh_width, mesh_height);
    cutilCheckMsg("execution failed\n");
}

#endif // #ifndef _SIMPLEGL_KERNEL_H_

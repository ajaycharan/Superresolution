#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_P_
#define _KERNEL_UPDATE_P_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_dualp(float *px, float *py, float *ux_, float *uy_, float epsilon_u, float sigma, float lambda,
                             unsigned int stride)
{


    //p^{n+1} = \frac{p^n + \sigma \nabla u^{n}}{1 + sigma*epsilon_u/ lambda}
    //p^{n+1} = p^{n+1} / max ( 1, |p^{n+1}|/lambda )

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    px[y*stride+x] = (px[y*stride+x] + sigma* ux_[y*stride+x])/(1+sigma*epsilon_u/lambda);
    py[y*stride+x] = (py[y*stride+x] + sigma* uy_[y*stride+x])/(1+sigma*epsilon_u/lambda);

    float pxval = px[y*stride+x];
    float pyval = py[y*stride+x];

    float reprojection = 0;
    reprojection   = sqrt(pxval*pxval + pyval*pyval);
    reprojection   = max(1.0f,reprojection/lambda);

    px[y*stride+x] = px[y*stride+x]/reprojection;
    py[y*stride+x] = py[y*stride+x]/reprojection;


}



// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_dual_variable_p(float *px, float *py, float* ux_, float *uy_, float epsilon_u, float sigma, float lambda,
                                              unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_dualp<<< grid, block>>>(px,py,ux_,uy_,epsilon_u, sigma, lambda, stride);
    cutilCheckMsg("execution failed\n");
}

#endif // #ifndef _SIMPLEGL_KERNEL_H_

#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_P_
#define _KERNEL_UPDATE_P_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_dualp(float *px, float *py, float *ux_, float *uy_, float epsilon_u, float sigma_p, float lambda,
                             unsigned int stride,unsigned int width)
{




    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex

    px[y*width+x] = (px[y*width+x] + sigma_p*lambda*ux_[y*width+x]);
    py[y*width+x] = (py[y*width+x] + sigma_p*lambda*uy_[y*width+x]);

    float pxval = px[y*width+x];
    float pyval = py[y*width+x];

    float reprojection = 0;
    reprojection   = sqrt(pxval*pxval + pyval*pyval);
    reprojection   = max(1.0f,reprojection);

    px[y*width+x] = px[y*width+x]/reprojection;
    py[y*width+x] = py[y*width+x]/reprojection;


//    px[y*stride+x] = (px[y*stride+x] + sigma_p*lambda*ux_[y*stride+x]);
//    py[y*stride+x] = (py[y*stride+x] + sigma_p*lambda*uy_[y*stride+x]);

//    float pxval = px[y*stride+x];
//    float pyval = py[y*stride+x];

//    float reprojection = 0;
//    reprojection   = sqrt(pxval*pxval + pyval*pyval);
//    reprojection   = max(1.0f,reprojection);

//    px[y*stride+x] = px[y*stride+x]/reprojection;
//    py[y*stride+x] = py[y*stride+x]/reprojection;


}



// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_dual_variable_p(float *px, float *py, float* ux_, float *uy_, float epsilon_u, float sigma_p, float lambda,
                                              unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(1, 1, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_dualp<<< grid, block>>>(px,py,ux_,uy_,epsilon_u, sigma_p, lambda, stride, mesh_width);
    cutilCheckMsg("execution failed\n");
}

#endif // #ifndef _SIMPLEGL_KERNEL_H_

#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_Q_
#define _KERNEL_UPDATE_Q_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *d_qi, int qStride,
                                                          float *d_DBWiu_fi, int DBWiu_fiStride,
                                                          float sigma_q,float xisqr,float epsilon_d,
                                                          int N_cols_low_img, int N_rows_low_img, int N_imgs)
{


    //q^{n+1} = \frac{q^n + \sigma \xi^{2} (DBWu_ - f)}{ 1 + epsilon_d*sigma_q/xisqr}
    // q^{n+1} =  max(-xisqr, min(xisqr, q^{n+1}))

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    if ( y*N_cols_low_img + x < N_cols_low_img*N_rows_low_img*N_imgs)
    {
       float result_val = d_qi[y*N_cols_low_img+x] + sigma_q*(d_DBWiu_fi[y*N_cols_low_img+x]);
       result_val = max(-1.0f, min(1.0f, result_val));
       d_qi[y*N_cols_low_img+x] = result_val;
    }


}


extern "C" void launch_kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *d_qi, int qStride,
                                                                 float *d_DBWiu_fi, int DBWiu_fiStride,
                                                                 float sigma_q,float xisqr,float epsilon_d,
                                                                 int N_cols_low_img, int N_rows_low_img, int N_imgs)
{
    dim3 block(2, 2, 1);
    dim3 grid(N_cols_low_img / block.x, N_rows_low_img*N_imgs / block.y, 1);
    kernel_q_SubtractDBWiu_fAdd_yAndReproject<<< grid, block>>>(d_qi, qStride,
                                                                d_DBWiu_fi, DBWiu_fiStride,
                                                                sigma_q,xisqr, epsilon_d,
                                                                N_cols_low_img, N_rows_low_img, N_imgs);
    cutilCheckMsg("execution failed\n");
}


#endif // #ifndef _SIMPLEGL_KERNEL_H_

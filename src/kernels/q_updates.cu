#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_Q_
#define _KERNEL_UPDATE_Q_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_dualq(int N_imgs, float** q, float** DBWu_, float epsilon_d, float sigma, float** f, float xisqr, unsigned int stride)
{


    //q^{n+1} = \frac{q^n + \sigma \xi^{2} (DBWu_ - f)}{ 1 + epsilon_d/}
    //q^{n+1} = p^{n+1} / max ( 1, |p^{n+1}|/lambda )

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    for (int img_no = 0 ; img_no < N_imgs; img_no++)
    {
        q[img_no][y*stride+x] = (q[img_no][y*stride+x] + sigma*xisqr*(DBWu_[img_no][y*stride+x] - f[img_no][y*stride+x]))/(1+sigma*epsilon_d/xisqr);
        q[img_no][y*stride+x] = (q[img_no][y*stride+x] + sigma*xisqr*(DBWu_[img_no][y*stride+x] - f[img_no][y*stride+x]))/(1+sigma*epsilon_d/xisqr);

        float pxval = q[img_no][y*stride+x];
        float pyval = q[img_no][y*stride+x];

        float reprojection = 0;
        reprojection   = max(-1.0f,min(1.0f,q[img_no][y*stride+x]));

        q[img_no][y*stride+x] = q[img_no][y*stride+x]/reprojection;
        q[img_no][y*stride+x] = q[img_no][y*stride+x]/reprojection;
    }


}



// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_dual_variable_q(int N_imgs, float** q, float** DBWu_, float epsilon_d, float sigma, float** f, float xisqr,
                                              unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_dualq<<< grid, block>>>(N_imgs, q, DBWu_, epsilon_d, sigma, f, xisqr, stride);
    cutilCheckMsg("execution failed\n");
}

#endif // #ifndef _SIMPLEGL_KERNEL_H_

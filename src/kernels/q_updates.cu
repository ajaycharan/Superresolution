#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_Q_
#define _KERNEL_UPDATE_Q_

#include <stdio.h>
#include <cutil_inline.h>

#include "cuPrintf.cu"

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *d_qi, int qStride,
                                                          float *d_DBWiu_fi, int DBWiu_fiStride,
                                                          float sigma_q,float xisqr,float epsilon_d,
                                                          int N_cols_low_img, int N_rows_low_img, int N_imgs)
{


    //q^{n+1} = \frac{q^n + \sigma \xi^{2} (DBWu_ - f)}{ 1 + epsilon_d*sigma_q/xisqr}
    // q^{n+1} =  max(-xisqr, min(xisqr, q^{n+1}))

    //unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    //unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

    // write output vertex
    if ( tid < N_cols_low_img*N_rows_low_img*N_imgs)
    {
        if(tid == 0)
        {
            cuPrintf("d_DBWiu_fi[tid]=%f\n",d_DBWiu_fi[tid]);
            cuPrintf("d_qi[tid]=%f\n", d_qi[tid]);
        }
       float result_val = d_qi[tid] + sigma_q*(d_DBWiu_fi[tid]);
//       result_val = max(-1.0f, min(1.0f, result_val));
       d_qi[tid] = result_val;
    }


}


extern "C" void launch_kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *d_qi, int qStride,
                                                                 float *d_DBWiu_fi, int DBWiu_fiStride,
                                                                 float sigma_q,float xisqr,float epsilon_d,
                                                                 int N_cols_low_img, int N_rows_low_img, int N_imgs)
{
    dim3 block(9, 1, 1);
    dim3 grid(N_cols_low_img*N_rows_low_img*N_imgs/9, 1);

    cudaPrintfInit();


    kernel_q_SubtractDBWiu_fAdd_yAndReproject<<< grid, block>>>(d_qi, qStride,
                                                                d_DBWiu_fi, DBWiu_fiStride,
                                                                sigma_q,xisqr, epsilon_d,
                                                                N_cols_low_img, N_rows_low_img, N_imgs);

    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    cutilCheckMsg("execution failed\n");
}


#endif // #ifndef _SIMPLEGL_KERNEL_H_

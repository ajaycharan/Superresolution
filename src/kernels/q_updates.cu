#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_Q_
#define _KERNEL_UPDATE_Q_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *result, int resultStride,
                                                          float *d_DBWiu,int DBWiuStride,
                                                          float *d_fi,   int imgStride,
                                                          float sigma_q,float xisqr,float epsilon_d,
                                                          int width_down, int height_down)
{


    //q^{n+1} = \frac{q^n + \sigma \xi^{2} (DBWu_ - f)}{ 1 + epsilon_d*sigma_q/xisqr}
    //q^{n+1} =  max(-1.0f, min(1.0f, q^{n+1}))

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    if ( y*resultStride + x < width_down*height_down)
    {

       float result_val = result[y*resultStride+x] + sigma_q*xisqr*(d_DBWiu[y*DBWiuStride+x]-d_fi[y*imgStride+x]);
       result_val = result_val/(1 + sigma_q*epsilon_d/xisqr);

       result_val = max(-xisqr*1.0f, min(xisqr, result_val)); // clamped reprojection

       result[y*resultStride+x] = result_val;
    }

}


extern "C" void launch_kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *result, int resultStride,
                                                                 float *d_DBWiu,int DBWiuStride,
                                                                 float *d_fi,   int imgStride,
                                                                 float sigma_q,float xisqr,float epsilon_d,
                                                                 int width_down, int height_down)
{
    dim3 block(8, 8, 1);
    dim3 grid(width_down / block.x, height_down / block.y, 1);
    kernel_q_SubtractDBWiu_fAdd_yAndReproject<<< grid, block>>>(result, resultStride,
                                                                d_DBWiu, DBWiuStride,
                                                                d_fi, imgStride,
                                                                sigma_q, xisqr, epsilon_d, width_down, height_down);//N_imgs, q, DBWu_, epsilon_d, sigma, f, xisqr, stride);
    cutilCheckMsg("execution failed\n");
}


#endif // #ifndef _SIMPLEGL_KERNEL_H_

#include "hip/hip_runtime.h"
#ifndef _KERNEL_UPDATE_Q_
#define _KERNEL_UPDATE_Q_

#include <stdio.h>
#include <cutil_inline.h>
#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif


__global__ void kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *d_qi, int qStride,
                                                          float *d_DBWiu_fi, int DBWiu_fiStride,
                                                          float sigma_q,float xisqr,float epsilon_d,
                                                          int width_down, int height_down)
{


    //q^{n+1} = \frac{q^n + \sigma \xi^{2} (DBWu_ - f)}{ 1 + epsilon_d*sigma_q/xisqr}
    // q^{n+1} =  max(-xisqr, min(xisqr, q^{n+1}))

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    if ( y*qStride + x < width_down*height_down)
    {

       float result_val = d_qi[y*qStride+x] + sigma_q*xisqr*(d_DBWiu_fi[y*DBWiu_fiStride+x]);
       result_val = result_val/(1 + sigma_q*epsilon_d/xisqr);

       result_val = max(-xisqr*1.0f, min(xisqr*1.0f, result_val)); // clamped reprojection
       //result_val = max(-1.0f, min(1.0f, result_val)); // to be L1 thing!
       d_qi[y*qStride+x] = result_val;
    }


}


extern "C" void launch_kernel_q_SubtractDBWiu_fAdd_yAndReproject(float *d_qi, int qStride,
                                                                 float *d_DBWiu_fi, int DBWiu_fiStride,
                                                                 float sigma_q,float xisqr,float epsilon_d,
                                                                 int width_down, int height_down)
{
    dim3 block(8, 8, 1);
    dim3 grid(width_down / block.x, height_down / block.y, 1);
    kernel_q_SubtractDBWiu_fAdd_yAndReproject<<< grid, block>>>(d_qi, qStride,
                                                                d_DBWiu_fi, DBWiu_fiStride,
                                                                sigma_q,xisqr, epsilon_d,
                                                                width_down, height_down);
    cutilCheckMsg("execution failed\n");
}


#endif // #ifndef _SIMPLEGL_KERNEL_H_
